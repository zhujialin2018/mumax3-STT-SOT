#include "hip/hip_runtime.h"
#include "stencil.h"

// Copy src (size S, larger) to dst (size D, smaller)
extern "C" __global__ void
copyunpad(float* __restrict__  dst, int Dx, int Dy, int Dz,
          float* __restrict__  src, int Sx, int Sy, int Sz) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix<Dx && iy<Dy && iz<Dz) {
        dst[index(ix, iy, iz, Dx, Dy, Dz)] = src[index(ix, iy, iz, Sx, Sy, Sz)];
    }
}

