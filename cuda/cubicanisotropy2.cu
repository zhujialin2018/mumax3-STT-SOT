#include "hip/hip_runtime.h"
#include "amul.h"
#include "float3.h"
#include <stdint.h>

// add cubic anisotropy field to B.
// B:      effective field in T
// m:      reduced magnetization (unit length)
// Ms:     saturation magnetization in A/m.
// K1:     Kc1 in J/m3
// K2:     Kc2 in T/m3
// C1, C2: anisotropy axes
//
// based on http://www.southampton.ac.uk/~fangohr/software/oxs_cubic8.html
extern "C" __global__ void
addcubicanisotropy2(float* __restrict__ Bx, float* __restrict__ By, float* __restrict__ Bz,
                    float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
                    float* __restrict__  Ms_, float  Ms_mul,
                    float* __restrict__  k1_, float  k1_mul,
                    float* __restrict__  k2_, float  k2_mul,
                    float* __restrict__  k3_, float  k3_mul,
                    float* __restrict__ c1x_, float c1x_mul,
                    float* __restrict__ c1y_, float c1y_mul,
                    float* __restrict__ c1z_, float c1z_mul,
                    float* __restrict__ c2x_, float c2x_mul,
                    float* __restrict__ c2y_, float c2y_mul,
                    float* __restrict__ c2z_, float c2z_mul,
                    int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {

        float invMs = inv_Msat(Ms_, Ms_mul, i);
        float  k1 = amul(k1_, k1_mul, i) * invMs;
        float  k2 = amul(k2_, k2_mul, i) * invMs;
        float  k3 = amul(k3_, k3_mul, i) * invMs;
        float3 u1 = normalized(vmul(c1x_, c1y_, c1z_, c1x_mul, c1y_mul, c1z_mul, i));
        float3 u2 = normalized(vmul(c2x_, c2y_, c2z_, c2x_mul, c2y_mul, c2z_mul, i));
        float3 u3 = cross(u1, u2); // 3rd axis perpendicular to u1,u2
        float3 m  = make_float3(mx[i], my[i], mz[i]);

        float u1m = dot(u1, m);
        float u2m = dot(u2, m);
        float u3m = dot(u3, m);

        float3 B = -2.0f*k1*((pow2(u2m) + pow2(u3m)) * (    (u1m) * u1) +
                             (pow2(u1m) + pow2(u3m)) * (    (u2m) * u2) +
                             (pow2(u1m) + pow2(u2m)) * (    (u3m) * u3))-
                   2.0f*k2*((pow2(u2m) * pow2(u3m)) * (    (u1m) * u1) +
                            (pow2(u1m) * pow2(u3m)) * (    (u2m) * u2) +
                            (pow2(u1m) * pow2(u2m)) * (    (u3m) * u3))-
                   4.0f*k3*((pow4(u2m) + pow4(u3m)) * (pow3(u1m) * u1) +
                            (pow4(u1m) + pow4(u3m)) * (pow3(u2m) * u2) +
                            (pow4(u1m) + pow4(u2m)) * (pow3(u3m) * u3));
        Bx[i] += B.x;
        By[i] += B.y;
        Bz[i] += B.z;
    }
}
