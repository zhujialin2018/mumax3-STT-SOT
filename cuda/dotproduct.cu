#include "hip/hip_runtime.h"

#include "float3.h"

// dst += prefactor * dot(a,b)
extern "C" __global__ void
dotproduct(float* __restrict__ dst, float prefactor,
           float* __restrict__ ax, float* __restrict__ ay, float* __restrict__ az,
           float* __restrict__ bx, float* __restrict__ by, float* __restrict__ bz,
           int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {
        float3 A = {ax[i], ay[i], az[i]};
        float3 B = {bx[i], by[i], bz[i]};
        dst[i] += prefactor * dot(A, B);
    }
}

