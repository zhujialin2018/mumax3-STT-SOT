#include "hip/hip_runtime.h"
#include <stdint.h>
#include "exchange.h"
#include "float3.h"
#include "stencil.h"
#include "amul.h"

// See exchange.go for more details.
extern "C" __global__ void
addexchange(float* __restrict__ Bx, float* __restrict__ By, float* __restrict__ Bz,
            float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
            float* __restrict__ Ms_, float Ms_mul,
            float* __restrict__ aLUT2d, uint8_t* __restrict__ regions,
            float wx, float wy, float wz, int Nx, int Ny, int Nz, uint8_t PBC) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }

    // central cell
    int I = idx(ix, iy, iz);
    float3 m0 = make_float3(mx[I], my[I], mz[I]);

    if (is0(m0)) {
        return;
    }

    uint8_t r0 = regions[I];
    float3 B  = make_float3(0.0,0.0,0.0);

    int i_;    // neighbor index
    float3 m_; // neighbor mag
    float a__; // inter-cell exchange stiffness

    // left neighbor
    i_  = idx(lclampx(ix-1), iy, iz);           // clamps or wraps index according to PBC
    m_  = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_  = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    a__ = aLUT2d[symidx(r0, regions[i_])];
    B += wx * a__ *(m_ - m0);

    // right neighbor
    i_  = idx(hclampx(ix+1), iy, iz);
    m_  = make_float3(mx[i_], my[i_], mz[i_]);
    m_  = ( is0(m_)? m0: m_ );
    a__ = aLUT2d[symidx(r0, regions[i_])];
    B += wx * a__ *(m_ - m0);

    // back neighbor
    i_  = idx(ix, lclampy(iy-1), iz);
    m_  = make_float3(mx[i_], my[i_], mz[i_]);
    m_  = ( is0(m_)? m0: m_ );
    a__ = aLUT2d[symidx(r0, regions[i_])];
    B += wy * a__ *(m_ - m0);

    // front neighbor
    i_  = idx(ix, hclampy(iy+1), iz);
    m_  = make_float3(mx[i_], my[i_], mz[i_]);
    m_  = ( is0(m_)? m0: m_ );
    a__ = aLUT2d[symidx(r0, regions[i_])];
    B += wy * a__ *(m_ - m0);

    // only take vertical derivative for 3D sim
    if (Nz != 1) {
        // bottom neighbor
        i_  = idx(ix, iy, lclampz(iz-1));
        m_  = make_float3(mx[i_], my[i_], mz[i_]);
        m_  = ( is0(m_)? m0: m_ );
        a__ = aLUT2d[symidx(r0, regions[i_])];
        B += wz * a__ *(m_ - m0);

        // top neighbor
        i_  = idx(ix, iy, hclampz(iz+1));
        m_  = make_float3(mx[i_], my[i_], mz[i_]);
        m_  = ( is0(m_)? m0: m_ );
        a__ = aLUT2d[symidx(r0, regions[i_])];
        B += wz * a__ *(m_ - m0);
    }

    float invMs = inv_Msat(Ms_, Ms_mul, I);
    Bx[I] += B.x*invMs;
    By[I] += B.y*invMs;
    Bz[I] += B.z*invMs;
}

