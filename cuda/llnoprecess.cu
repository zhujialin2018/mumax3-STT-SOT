#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"

// Landau-Lifshitz torque without precession
extern "C" __global__ void
llnoprecess(float* __restrict__  tx, float* __restrict__  ty, float* __restrict__  tz,
            float* __restrict__  mx, float* __restrict__  my, float* __restrict__  mz,
            float* __restrict__  hx, float* __restrict__  hy, float* __restrict__  hz, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {

        float3 m = {mx[i], my[i], mz[i]};
        float3 H = {hx[i], hy[i], hz[i]};

        float3 mxH = cross(m, H);
        float3 torque = -cross(m, mxH);

        tx[i] = torque.x;
        ty[i] = torque.y;
        tz[i] = torque.z;
    }
}

