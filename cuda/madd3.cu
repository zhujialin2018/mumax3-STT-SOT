
#include <hip/hip_runtime.h>

// dst[i] = fac1 * src1[i] + fac2 * src2[i] + fac3 * src3[i]
extern "C" __global__ void
madd3(float* __restrict__ dst,
      float* __restrict__ src1, float fac1,
      float* __restrict__ src2, float fac2,
      float* __restrict__ src3, float fac3, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;

    if(i < N) {
        dst[i] = (fac1 * src1[i]) + (fac2 * src2[i] + fac3 * src3[i]);
        // parens for better accuracy heun solver.
    }
}

