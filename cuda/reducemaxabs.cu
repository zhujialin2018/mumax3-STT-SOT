#include "hip/hip_runtime.h"
#include "reduce.h"
#include "atomicf.h"

#define load_fabs(i) fabs(src[i])

extern "C" __global__ void
reducemaxabs(float* __restrict__ src, float* __restrict__ dst, float initVal, int n) {
    reduce(load_fabs, fmax, atomicFmaxabs)
}

