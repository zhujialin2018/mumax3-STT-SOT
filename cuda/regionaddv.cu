
#include <hip/hip_runtime.h>
#include <stdint.h>

// add region-based vector to dst:
// dst[i] += LUT[region[i]]
extern "C" __global__ void
regionaddv(float* __restrict__ dstx, float* __restrict__ dsty, float* __restrict__ dstz,
           float* __restrict__ LUTx, float* __restrict__ LUTy, float* __restrict__ LUTz,
           uint8_t* regions, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {

        uint8_t r = regions[i];
        dstx[i] += LUTx[r];
        dsty[i] += LUTy[r];
        dstz[i] += LUTz[r];
    }
}

