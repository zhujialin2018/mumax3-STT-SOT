#include "hip/hip_runtime.h"
#include "stencil.h"

// shift dst by shy cells (positive or negative) along Z-axis.
// new edge value is clampL at left edge or clampR at right edge.
extern "C" __global__ void
shiftz(float* __restrict__  dst, float* __restrict__  src,
       int Nx,  int Ny,  int Nz, int shz, float clampL, float clampR) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if(ix < Nx && iy < Ny && iz < Nz) {
        int iz2 = iz-shz;
        float newval;
        if (iz2 < 0) {
            newval = clampL;
        } else if (iz2 >= Nz) {
            newval = clampR;
        } else {
            newval = src[idx(ix, iy, iz2)];
        }
        dst[idx(ix, iy, iz)] = newval;
    }
}

