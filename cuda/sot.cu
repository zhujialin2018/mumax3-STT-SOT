#include "hip/hip_runtime.h"
// Original implementation by Mykola Dvornik for mumax2
// Modified for mumax3 by Arne Vansteenkiste, 2013, 2016

#include <stdint.h>
#include "float3.h"
#include "constants.h"
#include "amul.h"

extern "C" __global__ void
addsotorque(float* __restrict__ tx, float* __restrict__ ty, float* __restrict__ tz,
                      float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
                      float* __restrict__ Ms_,        float  Ms_mul,
                      float* __restrict__ jz_,        float  jz_mul,
                      float* __restrict__ px_,        float  px_mul,
                      float* __restrict__ py_,        float  py_mul,
                      float* __restrict__ pz_,        float  pz_mul,
                      float* __restrict__ alpha_,     float  alpha_mul,
                      float* __restrict__ spinhall_,  float  spinhall_mul,
                      float* __restrict__ hfloverhdl_,float  hfloverhdl_mul,
                      float* __restrict__ thickness_, float  thickness_mul,
                      int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {

        float3 m = make_float3(mx[i], my[i], mz[i]);
        float  J = amul(jz_, jz_mul, i);
        float3 p = normalized(vmul(px_, py_, pz_, px_mul, py_mul, pz_mul, i));
        float  Ms           = amul(Ms_, Ms_mul, i);
        float  alpha        = amul(alpha_, alpha_mul, i);
        float  spinhall     = amul(spinhall_, spinhall_mul, i);
        float  hfloverhdl   = amul(hfloverhdl_, hfloverhdl_mul, i);

        float thickness = amul(thickness_, thickness_mul, i);

        if (J == 0.0f || Ms == 0.0f) {
            return;
        }

        float beta    = (HBAR / QE) * (spinhall * J / (2.0 * thickness * Ms) );

        float B = beta * hfloverhdl;

        float gilb     = 1.0f / (1.0f + alpha * alpha);
        float mxpxmFac = gilb * (beta + alpha * B);
        float pxmFac   = gilb * (B - alpha * beta);

        float3 pxm      = cross(p, m);
        float3 mxpxm    = cross(m, pxm);

        tx[i] += mxpxmFac * mxpxm.x + pxmFac * pxm.x;
        ty[i] += mxpxmFac * mxpxm.y + pxmFac * pxm.y;
        tz[i] += mxpxmFac * mxpxm.z + pxmFac * pxm.z;
    }
}

